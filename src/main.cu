#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include <functions.cuh>
#include <kernels.cuh>

using std::cout;
using std::cin;
using std::endl;

void generateFunction(uint16_t *d_out, uint64_t size, uint64_t i0);

uint64_t findMaximum(const uint64_t *data, uint64_t size);

uint64_t generateAndFindMaximum(uint16_t *d_fn, uint64_t *d_buf, uint64_t *h_out, uint64_t size, uint64_t i0);

int main()
{
    const uint64_t size = 1ULL << 31u;

    uint16_t maxN = 1;
    uint64_t maxArg = 2;

    uint16_t *d_fn;
    hipMalloc(&d_fn, size * sizeof(*d_fn));

    // oh my god this is so wrong
    uint64_t blocks = (size + Kernels::numReduceThreads - 1) / Kernels::numReduceThreads;
    uint64_t *d_buf;
    hipMalloc(&d_buf, blocks * sizeof(*d_buf));

    auto *h_out = new uint64_t[blocks];

    for (int i = 0; i < 32; i++)
    {
        auto newMax = generateAndFindMaximum(d_fn, d_buf, h_out, size, i * size);
        uint16_t newMaxN = newMax & 0xffffU;
        uint64_t newMaxArg = newMax >> 16U;
        if (newMaxN > maxN)
        {
            maxN = newMaxN;
            maxArg = newMaxArg;
        }
    }

    hipFree(d_fn);
    hipFree(d_buf);
    delete [] h_out;

    cout << "Final max = " << maxArg << ", " << maxN << endl;

    return 0;
}

void generateFunction(uint16_t *d_out, uint64_t size, uint64_t i0)
{
    int threads = Kernels::numGenThreads;
    uint64_t blocks = (size + threads - 1) / threads;
    Kernels::genFnKn<<<blocks, threads>>>(d_out, size, i0);
}

uint64_t findMaximum(const uint64_t *data, uint64_t size)
{
    uint64_t max = data[0];
    for (uint64_t i = 0; i < size; i++)
    {
        if ((data[i] & 0xffffu) > (max & 0xffffu))
        {
            max = data[i];
        }
    }
    return max;
}

uint64_t generateAndFindMaximum(uint16_t *d_fn, uint64_t *d_buf, uint64_t *h_out, uint64_t size, uint64_t i0)
{
    cout << "Generating " << size << " values from " << i0 << endl;
    generateFunction(d_fn, size, i0);

//    hipDeviceSynchronize();
    cout << "Looking for maximum" << endl;

    // oh my god this is so wrong
    uint64_t blocks = (size + Kernels::numReduceThreads - 1) / Kernels::numReduceThreads;

    Kernels::findMax<<<blocks, Kernels::numReduceThreads>>>(d_buf, d_fn, size);

    hipMemcpy(h_out, d_buf, blocks * sizeof(*d_buf), hipMemcpyDeviceToHost);

    auto max = findMaximum(h_out, blocks);
    uint16_t maxN = max & 0xffffu;
    uint64_t maxArg = (max >> 16U) + i0;

    cout << "Maximum = " << maxArg << ", " << maxN << endl;

    return (maxArg << 16U) | maxN;
}
