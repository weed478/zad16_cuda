#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>

#include <functions.cuh>
#include <kernels.cuh>

using std::cout;
using std::cin;
using std::endl;

void generateFunction(uint16_t *d_out, uint64_t size, uint64_t i0);

uint64_t findMaximum(const uint64_t *data, uint64_t size);

uint64_t generateAndFindMaximum(uint16_t *d_fn, uint64_t *d_buf, uint64_t size, uint64_t i0);

// max in 2^31 * 1024: 2156795915823, 1419

int main()
{
    size_t freeMem = 0;
    hipMemGetInfo(&freeMem, nullptr);

    freeMem -= freeMem / 10;
    cout << "Using " << freeMem / (1u << 20u) << " MB" << endl;

    const uint64_t size = freeMem / 2;

    uint16_t maxN = 1;
    uint64_t maxArg = 2;

    uint16_t *d_fn;
    hipMalloc(&d_fn, size * sizeof(*d_fn));

    uint64_t blocks = ((size + 2ull - 1ull) / 2ull + Kernels::numReduceThreads - 1ull) / Kernels::numReduceThreads;
    uint64_t *d_buf;
    hipHostMalloc(&d_buf, blocks * sizeof(*d_buf));

    for (int i = 0; i < 4; i++)
    {
        auto newMax = generateAndFindMaximum(d_fn, d_buf, size, i * size);
        uint16_t newMaxN = newMax & 0xffffU;
        uint64_t newMaxArg = newMax >> 16U;
        if (newMaxN > maxN)
        {
            maxN = newMaxN;
            maxArg = newMaxArg;
        }
    }

    hipFree(d_fn);
    hipFree(d_buf);

    cout << "Final max = " << maxArg << ", " << maxN << endl;

    return 0;
}

void generateFunction(uint16_t *d_out, uint64_t size, uint64_t i0)
{
    int threads = Kernels::numGenThreads;
    uint64_t blocks = (size + threads - 1ull) / threads;
    Kernels::genFnKn<<<blocks, threads>>>(d_out, size, i0);
}

uint64_t findMaximum(const uint64_t *data, uint64_t size)
{
    uint64_t max = data[0];
    for (uint64_t i = 0; i < size; i++)
    {
        if ((data[i] & 0xffffu) > (max & 0xffffu))
        {
            max = data[i];
        }
    }
    return max;
}

uint64_t generateAndFindMaximum(uint16_t *d_fn, uint64_t *d_buf, uint64_t size, uint64_t i0)
{
    cout << "Generating " << size << " values from " << i0 << endl;
    generateFunction(d_fn, size, i0);

    uint64_t blocks = ((size + 2ull - 1ull) / 2ull + Kernels::numReduceThreads - 1ull) / Kernels::numReduceThreads;
    Kernels::findMax<<<blocks, Kernels::numReduceThreads>>>(d_buf, d_fn, size);

    hipDeviceSynchronize();
    auto max = findMaximum(d_buf, blocks);
    uint16_t maxN = max & 0xffffu;
    uint64_t maxArg = (max >> 16U) + i0;

    cout << "Maximum = " << maxArg << ", " << maxN << endl;

    return (maxArg << 16U) | maxN;
}
