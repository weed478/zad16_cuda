#include "hip/hip_runtime.h"
#include <kernels.cuh>
#include <functions.cuh>

__global__
void Kernels::genFnKn(uint16_t *out, uint64_t size, uint64_t i0)
{
    uint64_t i = (uint64_t)blockIdx.x * (uint64_t)blockDim.x + (uint64_t)threadIdx.x;
    if (1 < i && i < size)
        out[i] = getNForSeries(i0 + i);
}

__global__ void Kernels::findMax(uint64_t *out, const uint16_t *in, uint64_t size)
{
    __shared__ uint64_t sdata[numReduceThreads];

    const int stride = numReduceThreads;

    unsigned int t = threadIdx.x;
    uint64_t i = (uint64_t)blockIdx.x * (uint64_t)blockDim.x * 2 + (uint64_t)threadIdx.x;

    if (i < size)
    {
        if (i + stride < size && in[i + stride] > in[i])
            sdata[t] = ((i + stride) << 16u) | in[i + stride];
        else
            sdata[t] = (i << 16u) | in[i];
    }

    __syncthreads();

    for (unsigned int s = stride / 2; s > 0; s >>= 1u)
    {
        if (t < s)
        {
            if ((sdata[t] & 0xffffu) > (sdata[t + s] & 0xffffu))
                sdata[t] = sdata[t];
            else
                sdata[t] = sdata[t + s];
        }
        __syncthreads();
    }

    if (t == 0)
        out[blockIdx.x] = sdata[0];
}


